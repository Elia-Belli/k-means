#include "hip/hip_runtime.h"
/*
 * k-Means clustering algorithm
 *
 * CUDA version
 *
 * Parallel computing (Degree in Computer Engineering)
 * 2022/2023
 *
 * Version: 1.0
 *
 * (c) 2022 Diego García-Álvarez, Arturo Gonzalez-Escribano
 * Grupo Trasgo, Universidad de Valladolid (Spain)
 *
 * This work is licensed under a Creative Commons Attribution-ShareAlike 4.0 International License.
 * https://creativecommons.org/licenses/by-sa/4.0/
 */
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <float.h>
#include <hip/hip_runtime.h>


#define MAXLINE 2000
#define MAXCAD 200

//Macros
#define MIN(a,b) ((a) < (b) ? (a) : (b))
#define MAX(a,b) ((a) > (b) ? (a) : (b))

void getDeviceProperties(int device, int* SMcount, int* maxThreadsPerBlock, size_t* maxSharedMem);
int getBlockSize(int threads, size_t sharedPerThread, int regsPerThread);

/*
 * Macros to show errors when calling a CUDA library function,
 * or after launching a kernel
 */
#define CHECK_CUDA_CALL( a )	{ \
	hipError_t ok = a; \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA call in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}
#define CHECK_CUDA_LAST()	{ \
	hipError_t ok = hipGetLastError(); \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA last in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}

__device__
static float atomicMax(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__device__
float euclideanDistance(float *point, float *center, int samples)
{
	float dist = 0.0;
	for(int i = 0; i < samples; i++) 
	{
		dist += (point[i]-center[i])*(point[i]-center[i]);
	}

	dist = sqrt(dist);
	return dist;
}


__global__ void kmeansMapClass(float *data, float *centroids, int *classMap, int *pointsPerClass,
            int* changes, int lines, int samples, int K)
{
    int globID = blockIdx.x * blockDim.x + threadIdx.x;
    int locID = threadIdx.x;
    int i;

    extern __shared__ int shared[];  
    int* localPointsPerClass = (int*) &shared[0];
    int* localChanges = (int*) &shared[K];
    float* localCentroids = (float*) &shared[K+1]; 

    float minDist = FLT_MAX, dist;
    int cluster = 1;

    // Init shared pointsPerClass
    for(i = locID; i < K; i += blockDim.x)
    {
        localPointsPerClass[i] = 0;
    }

    // Copy centroids in SharedMem to reduce global memory access
    for(i = locID; i < K*samples; i += blockDim.x)
    {
        localCentroids[i] = centroids[i];
    }

    if(locID == 0) localChanges[0] = 0;

    __syncthreads();

    if(globID < lines)
    {
        for(i = 0; i < K; i++)
        {
            dist = euclideanDistance(&data[globID * samples], &localCentroids[i * samples], samples);

            if(dist < minDist)
            {
                minDist = dist;
                cluster = i+1;
            }
        }

        if(classMap[globID] != cluster)
        {
            atomicAdd(localChanges, 1);
            classMap[globID] = cluster;
        }

        atomicAdd(&localPointsPerClass[cluster-1], 1);

    }

    __syncthreads();

    atomicAdd(changes, *localChanges);

    // Sum localPointPerClass in global pointsPerClass
    for(i = locID; i < K; i += blockDim.x)
    {
        atomicAdd(&pointsPerClass[i], localPointsPerClass[i]);
    }
}

__global__ void kmeansMapClassTiling(float *data, float *centroids, int *classMap, int *pointsPerClass,
            int* changes, int lines, int samples, int K)
{
    int globID = blockIdx.x * blockDim.x + threadIdx.x;
    int locID = threadIdx.x;

    extern __shared__ int shared[];
    int *localPointsPerClass = (int*) &shared[0];
    int *localChanges = (int*) &shared[K];
    float *centroid = (float*) &shared[K+1];

    float minDist = FLT_MAX, dist;
    int i, j, cluster = 1;

    // Init shared pointsPerClass
    for(i = locID; i < K; i += blockDim.x)
    {
        localPointsPerClass[i] = 0;
    }

    if(locID == 0) 
    {
        *localChanges = 0;
    }

    
    for(i = 0; i < K; i++)
    {   
        
        for(j = locID; j < samples; j+=blockDim.x)
        {
            centroid[j] = centroids[i*samples + j];
        }
        __syncthreads();

        if(globID < lines)
        {
            dist = euclideanDistance(&data[globID * samples], &centroids[i * samples], samples);

            if(dist < minDist)
            {
                minDist = dist;
                cluster = i+1;
            }
        }
        __syncthreads();

    }

    if(globID < lines)
    {
        if(classMap[globID] != cluster)
        {
            atomicAdd(localChanges, 1);
            classMap[globID] = cluster;
        }

        atomicAdd(&localPointsPerClass[cluster-1], 1);

    }

    __syncthreads();

    if(locID == 0)
    {
        atomicAdd(changes, *localChanges);
    }

    for(i = locID; i < K; i+=blockDim.x)
    {
        atomicAdd(&pointsPerClass[i], localPointsPerClass[i]);
    }
}


// 2. 
__global__ void kmeansCentroidsSum(float *data, float *auxCentroids, int *pointPerClass, int *classMap,
                            int lines, int samples, int K)
{   
    int globID = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = gridDim.x * blockDim.x;
    int i, cluster;

    if(globID < lines)
    {
        cluster = classMap[globID] - 1;
        for(i = 0; i < samples; i++)
        {
            atomicAdd(&auxCentroids[cluster * samples + i], data[globID * samples + i]);
        }
    }
}

// 3. Before we summed directly data[]/pointsPerClass[] in auxCentroids[]
//      but doing so requires (lines*samples) divisions
//    Since K << lines, doing the division after the sum reduces the number of div
__global__ void kmeansCentroidsDiv(float* auxCentroids, int* pointsPerClass, int samples, int K)
{
    int globID = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = gridDim.x * blockDim.x;
    int i;

    for(i = globID; i < K*samples; i += gridSize)
    {
        auxCentroids[i] /= pointsPerClass[i/samples];
    } 
}

__global__ void kmeansMaxDist(float *auxCentroids, float* centroids, int* pointPerClass, 
                                float* maxDist, int samples, int K)
{   
    int globID = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = gridDim.x * blockDim.x;

    __shared__ float localMaxDist;
    
    int i;
    float dist;

    if(globID == 0) 
        localMaxDist = 0;

    __syncthreads();

    for(i = globID; i < K; i += gridSize)
    {
        dist = euclideanDistance(&auxCentroids[i * samples], &centroids[i * samples], samples);
        atomicMax(&localMaxDist, dist);
    }

    __syncthreads();
    if(globID == 0) 
        atomicMax(maxDist, localMaxDist);
    
}


/* 
Function showFileError: It displays the corresponding error during file reading.
*/
void showFileError(int error, char* filename)
{
	printf("Error\n");
	switch (error)
	{
		case -1:
			fprintf(stderr,"\tFile %s has too many columns.\n", filename);
			fprintf(stderr,"\tThe maximum number of columns has been exceeded. MAXLINE: %d.\n", MAXLINE);
			break;
		case -2:
			fprintf(stderr,"Error reading file: %s.\n", filename);
			break;
		case -3:
			fprintf(stderr,"Error writing file: %s.\n", filename);
			break;
	}
	fflush(stderr);	
}

/* 
Function readInput: It reads the file to determine the number of rows and columns.
*/
int readInput(char* filename, int *lines, int *samples)
{
    FILE *fp;
    char line[MAXLINE] = "";
    char *ptr;
    const char *delim = "\t";
    int contlines, contsamples = 0;
    
    contlines = 0;

    if ((fp=fopen(filename,"r"))!=NULL)
    {
        while(fgets(line, MAXLINE, fp)!= NULL) 
		{
			if (strchr(line, '\n') == NULL)
			{
				return -1;
			}
            contlines++;       
            ptr = strtok(line, delim);
            contsamples = 0;
            while(ptr != NULL)
            {
            	contsamples++;
				ptr = strtok(NULL, delim);
	    	}	    
        }
        fclose(fp);
        *lines = contlines;
        *samples = contsamples;  
        return 0;
    }
    else
	{
    	return -2;
	}
}

/* 
Function readInput2: It loads data from file.
*/
int readInput2(char* filename, float* data)
{
    FILE *fp;
    char line[MAXLINE] = "";
    char *ptr;
    const char *delim = "\t";
    int i = 0;
    
    if ((fp=fopen(filename,"rt"))!=NULL)
    {
        while(fgets(line, MAXLINE, fp)!= NULL)
        {         
            ptr = strtok(line, delim);
            while(ptr != NULL)
            {
            	data[i] = atof(ptr);
            	i++;
				ptr = strtok(NULL, delim);
	   		}
	    }
        fclose(fp);
        return 0;
    }
    else
	{
    	return -2; //No file found
	}
}

/* 
Function writeResult: It writes in the output file the cluster of each sample (point).
*/
int writeResult(int *classMap, int lines, const char* filename)
{	
    FILE *fp;
    
    if ((fp=fopen(filename,"wt"))!=NULL)
    {
        for(int i=0; i<lines; i++)
        {
        	fprintf(fp,"%d\n",classMap[i]);
        }
        fclose(fp);  
   
        return 0;
    }
    else
	{
    	return -3; //No file found
	}
}

/*

Function initCentroids: This function copies the values of the initial centroids, using their 
position in the input data structure as a reference map.
*/
void initCentroids(const float *data, float* centroids, int* centroidPos, int samples, int K)
{
	int i;
	int idx;
	for(i=0; i<K; i++)
	{
		idx = centroidPos[i];
		memcpy(&centroids[i*samples], &data[idx*samples], (samples*sizeof(float)));
	}
}

/*
Function euclideanDistance: Euclidean distance
This function could be modified
*/
float euclideanDistanceCPU(float *point, float *center, int samples)
{
	float dist=0.0;
	for(int i=0; i<samples; i++) 
	{
		dist+= (point[i]-center[i])*(point[i]-center[i]);
	}
	dist = sqrt(dist);
	return(dist);
}


/*
Function zeroFloatMatriz: Set matrix elements to 0
This function could be modified
*/
void zeroFloatMatriz(float *matrix, int rows, int columns)
{
	int i,j;
	for (i=0; i<rows; i++)
		for (j=0; j<columns; j++)
			matrix[i*columns+j] = 0.0;	
}

/*
Function zeroIntArray: Set array elements to 0
This function could be modified
*/
void zeroIntArray(int *array, int size)
{
	int i;
	for (i=0; i<size; i++)
		array[i] = 0;	
}



int main(int argc, char* argv[])
{

	//START CLOCK***************************************
	double start, end;
	start = clock();
	//**************************************************
	/*
	* PARAMETERS
	*
	* argv[1]: Input data file
	* argv[2]: Number of clusters
	* argv[3]: Maximum number of iterations of the method. Algorithm termination condition.
	* argv[4]: Minimum percentage of class changes. Algorithm termination condition.
	*          If between one iteration and the next, the percentage of class changes is less than
	*          this percentage, the algorithm stops.
	* argv[5]: Precision in the centroid distance after the update.
	*          It is an algorithm termination condition. If between one iteration of the algorithm 
	*          and the next, the maximum distance between centroids is less than this precision, the
	*          algorithm stops.
	* argv[6]: Output file. Class assigned to each point of the input file.
	* */
	if(argc !=  7)
	{
		fprintf(stderr,"EXECUTION ERROR K-MEANS: Parameters are not correct.\n");
		fprintf(stderr,"./KMEANS [Input Filename] [Number of clusters] [Number of iterations] [Number of changes] [Threshold] [Output data file]\n");
		fflush(stderr);
		exit(-1);
	}

	// Reading the input data
	// lines = number of points; samples = number of dimensions per point
	int lines = 0, samples= 0;  
	
	int error = readInput(argv[1], &lines, &samples);
	if(error != 0)
	{
		showFileError(error,argv[1]);
		exit(error);
	}
	
	float *data = (float*)calloc(lines*samples,sizeof(float));
	if (data == NULL)
	{
		fprintf(stderr,"Memory allocation error.\n");
		exit(-4);
	}
	error = readInput2(argv[1], data);
	if(error != 0)
	{
		showFileError(error,argv[1]);
		exit(error);
	}

	// Parameters
	int K=atoi(argv[2]); 
	int maxIterations=atoi(argv[3]);
	int minChanges= (int)(lines*atof(argv[4])/100.0);
	float maxThreshold=atof(argv[5]);

	int *centroidPos = (int*)calloc(K,sizeof(int));
	float *centroids = (float*)calloc(K*samples,sizeof(float));
	int *classMap = (int*)calloc(lines,sizeof(int));

    if (centroidPos == NULL || centroids == NULL || classMap == NULL)
	{
		fprintf(stderr,"Memory allocation error.\n");
		exit(-4);
	}

	// Initial centrodis
	srand(0);
	int i;
	for(i=0; i<K; i++) 
		centroidPos[i]=rand()%lines;
	
	// Loading the array of initial centroids with the data from the array data
	// The centroids are points stored in the data array.
	initCentroids(data, centroids, centroidPos, samples, K);

	#ifdef DEBUG
		printf("\n\tData file: %s \n\tPoints: %d\n\tDimensions: %d\n", argv[1], lines, samples);
		printf("\tNumber of clusters: %d\n", K);
		printf("\tMaximum number of iterations: %d\n", maxIterations);
		printf("\tMinimum number of changes: %d [%g%% of %d points]\n", minChanges, atof(argv[4]), lines);
		printf("\tMaximum centroid precision: %f\n", maxThreshold);
	#endif
	
	//END CLOCK*****************************************
    #ifdef DEBUG
		end = clock();
		printf("\nMemory allocation: %f seconds\n", (double)(end - start) / CLOCKS_PER_SEC);
		fflush(stdout);
    #endif

	CHECK_CUDA_CALL( hipSetDevice(0) );
	CHECK_CUDA_CALL( hipDeviceSynchronize() );
	//**************************************************
	//START CLOCK***************************************
	start = clock();
	//**************************************************
	char *outputMsg = (char *)calloc(10000,sizeof(char));
	char line[100];

	// int j;
	// int classe;
	// float dist, minDist;
	int it=1;
	int changes = 0;
	float maxDist;

	//pointPerClass: number of points classified in each class
	//auxCentroids: mean of the points in each class
	int *pointsPerClass = (int *)malloc(K*sizeof(int));
	float *auxCentroids = (float*)malloc(K*samples*sizeof(float));
	float *distCentroids = (float*)malloc(K*sizeof(float)); 
	if (pointsPerClass == NULL || auxCentroids == NULL || distCentroids == NULL)
	{
		fprintf(stderr,"Memory allocation error.\n");
		exit(-4);
	}

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 *
 */
    int SMcount, maxThreadsPerBlock;
    size_t maxSharedMem, sharedMapClassBase, sharedMapClassTiling, sharedMapClass;
    getDeviceProperties(0, &SMcount, &maxThreadsPerBlock, &maxSharedMem);

    sharedMapClassTiling = (K+1+samples) * sizeof(int) ;
    sharedMapClassBase = (K+1) * sizeof(int) + (K*samples) * sizeof(float);

    float *d_data, *d_centroids, *d_auxCentroids, *d_maxDist;
    int *d_classMap, *d_changes, *d_pointPerClass;
    int anotherIteration = 1;

    int gridSize, blockSize; 
    blockSize = getBlockSize(lines, sharedMapClass, 32);
    gridSize = ceil(lines/blockSize);

    #ifdef DEBUG
    printf("\nBlockSize: %d\n\n", blockSize);
    #endif

    // Allocation of GPU data structures
    CHECK_CUDA_CALL(hipMalloc((void**) &d_data, lines*samples*sizeof(float)));  
    CHECK_CUDA_CALL(hipMalloc((void**) &d_centroids, K*samples*sizeof(float)));     
    CHECK_CUDA_CALL(hipMalloc((void**) &d_auxCentroids, K*samples*sizeof(float)));  
    CHECK_CUDA_CALL(hipMalloc((void**) &d_classMap, lines*sizeof(int)));    
    CHECK_CUDA_CALL(hipMalloc((void**) &d_pointPerClass, K*sizeof(int)));    
    CHECK_CUDA_CALL(hipMalloc((void**) &d_maxDist, sizeof(float))); 
    CHECK_CUDA_CALL(hipMalloc((void**) &d_changes, sizeof(int)));   

    // Send data and initial centroids to GPU
    CHECK_CUDA_CALL(hipMemcpy(d_data, data, lines*samples*sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_CALL(hipMemcpy(d_centroids, centroids, K*samples*sizeof(float), hipMemcpyHostToDevice));
    // Initialize ClassMap on GPU
    CHECK_CUDA_CALL(hipMemset(d_classMap, 0, lines*sizeof(int)));
    
    // Kernel Arguments
    void* argsMapClass[] = {&d_data, &d_centroids, &d_classMap, &d_pointPerClass, &d_changes, &lines, &samples, &K};
    void* argsCentroidsSum[] = {&d_data, &d_auxCentroids, &d_pointPerClass, &d_classMap, &lines, &samples, &K};
    void* argsCentroidsDiv[] = {&d_auxCentroids, &d_pointPerClass, &samples, &K};
    void* argsMaxDist[] = {&d_auxCentroids, &d_centroids, &d_pointPerClass, &d_maxDist, &samples, &K};

	do{
        // Initialize MaxDist & Changes on GPU
        CHECK_CUDA_CALL(hipMemset(d_changes, 0, sizeof(int)));
        CHECK_CUDA_CALL(hipMemset(d_maxDist, FLT_MIN, sizeof(float)));
        CHECK_CUDA_CALL(hipMemset(d_auxCentroids, 0.0, K*samples*sizeof(float)));
        CHECK_CUDA_CALL(hipMemset(d_pointPerClass, 0, K*sizeof(int)));

        // Kernerls
        if(sharedMapClassBase < maxSharedMem)
        {
            CHECK_CUDA_CALL(hipLaunchKernel((void*) kmeansMapClass, gridSize, blockSize, argsMapClass, sharedMapClassBase, NULL));
        }else
        {
            CHECK_CUDA_CALL(hipLaunchKernel((void*) kmeansMapClassTiling, gridSize, blockSize, argsMapClass, sharedMapClassTiling, NULL));
        }
        CHECK_CUDA_CALL(hipDeviceSynchronize());

        CHECK_CUDA_CALL(hipLaunchKernel((void*) kmeansCentroidsSum, gridSize, blockSize, argsCentroidsSum, 0, NULL));
        CHECK_CUDA_CALL(hipDeviceSynchronize());

        CHECK_CUDA_CALL(hipLaunchKernel((void*) kmeansCentroidsDiv, gridSize, blockSize, argsCentroidsDiv, 0, NULL));
        CHECK_CUDA_CALL(hipDeviceSynchronize());

        CHECK_CUDA_CALL(hipLaunchKernel((void*) kmeansMaxDist, gridSize, blockSize, argsMaxDist, sizeof(float), NULL));
        CHECK_CUDA_CALL(hipDeviceSynchronize());

        // Get MaxDist & Changes back to CPU
        CHECK_CUDA_CALL(hipMemcpy(&maxDist, d_maxDist, sizeof(float), hipMemcpyDeviceToHost));
        CHECK_CUDA_CALL(hipMemcpy(&changes, d_changes, sizeof(int), hipMemcpyDeviceToHost));

        // Print iteration info
        #ifdef DEBUG
        sprintf(line, "\n[%d] Cluster changes: %d\tMax. centroid distance: %f", it, changes, maxDist);
        outputMsg = strcat(outputMsg, line);
        #endif

        // Check Termination Conditions
        anotherIteration = (changes > minChanges) && (it < maxIterations) && (maxDist > maxThreshold);

        if(anotherIteration){
            // Update Centroids for the next iteration
            CHECK_CUDA_CALL(hipMemcpy(d_centroids, d_auxCentroids, K*samples*sizeof(float), hipMemcpyDeviceToDevice));    
            it++;
        }
        
    } while(anotherIteration);

    // Copy final ClassMap on CPU
    CHECK_CUDA_CALL(hipMemcpy(classMap, d_classMap, lines*sizeof(int), hipMemcpyDeviceToHost));

    // Free GPU memory
    CHECK_CUDA_CALL(hipFree(d_pointPerClass));
    CHECK_CUDA_CALL(hipFree(d_classMap));
    CHECK_CUDA_CALL(hipFree(d_centroids)); 
    CHECK_CUDA_CALL(hipFree(d_auxCentroids));
    CHECK_CUDA_CALL(hipFree(d_data));
    CHECK_CUDA_CALL(hipFree(d_maxDist));
    CHECK_CUDA_CALL(hipFree(d_changes));

/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */
	// Output and termination conditions

	CHECK_CUDA_CALL( hipDeviceSynchronize() );

	//END CLOCK*****************************************
	end = clock();
    #ifdef DEBUG
		printf("%s",outputMsg);
		printf("\nComputation: %f seconds", (double)(end - start) / CLOCKS_PER_SEC);
		if (changes <= minChanges) {
			printf("\n\nTermination condition:\nMinimum number of changes reached: %d [%d]", changes, minChanges);
		}
		else if (it >= maxIterations) {
			printf("\n\nTermination condition:\nMaximum number of iterations reached: %d [%d]", it, maxIterations);
		}
		else {
			printf("\n\nTermination condition:\nCentroid update precision reached: %g [%g]", maxDist, maxThreshold);
		}
    #else
        printf("cuda,%f", (double)(end - start) / CLOCKS_PER_SEC);
	#endif
	fflush(stdout);
	//**************************************************
	//START CLOCK***************************************
	start = clock();
	//**************************************************

	// Writing the classification of each point to the output file.
	error = writeResult(classMap, lines, argv[6]);
	if(error != 0)
	{
		showFileError(error, argv[6]);
		exit(error);
	}

	//Free memory
	free(data);
	free(classMap);
	free(centroidPos);
	free(centroids);
	free(distCentroids);
	free(pointsPerClass);
	free(auxCentroids);

	//END CLOCK*****************************************
    #ifdef DEBUG
	end = clock();
	printf("\n\nMemory deallocation: %f seconds\n",(double)(end - start) / CLOCKS_PER_SEC);
	fflush(stdout);
    #endif
	//***************************************************/
	return 0;
}


/*
    Gets properties of cuda device
    in: int device (number of device)
    out: int* SMcount, int* maxSharedMem
*/
void getDeviceProperties(int device, int* SMcount, int* threadsPerBlock, size_t* maxSharedMem)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);

    *SMcount = prop.multiProcessorCount;
    *threadsPerBlock = prop.maxThreadsPerBlock;
    *maxSharedMem = prop.sharedMemPerBlock;

    #ifdef DEBUG
    printf("  Memory Clock Rate (MHz): %d\n", prop.memoryClockRate/1024);
    printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);

    printf("  Peak Memory Bandwidth (GB/s): %.1f\n",
        2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    printf("  Total global memory (Gbytes) %.1f\n",(float)(prop.totalGlobalMem)/1024.0/1024.0/1024.0);
    printf("  Shared memory per block (Bytes) %.1f\n",(float)(prop.sharedMemPerBlock));
    printf("  Shared memory per SM (Bytes) %.1f\n",(float)(prop.sharedMemPerMultiprocessor));
    
    printf("  SM count : %d\n", prop.multiProcessorCount);
    printf("  Warp-size: %d\n", prop.warpSize);
    printf("  max-grid-size: (%d, %d, %d)\n", prop.maxGridSize[0],prop.maxGridSize[1],prop.maxGridSize[2]);
    printf("  max-threads-per-block: %d\n", prop.maxThreadsPerBlock);
    printf("  max-threads-per-multiprocessor: %d\n", prop.maxThreadsPerMultiProcessor);
    printf("  register-per-block: %d\n", prop.regsPerBlock);
    #endif
    
}

int getBlockSize(int threads, size_t sharedPerThread, int regsPerThread)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    int warpSize = prop.warpSize;
    int regsPerBlock = prop.regsPerBlock;
    int sharedMemory = prop.sharedMemPerBlock;

    // For cc >= 3.0 we have at least 4 warpSchedulers per SM

    /* Registers per Thread for each kernel
    - kmeansMapClass : 32
    - kmeansMapClassOptimized : 30
    - kmeansCentroidsSum : 18
    - kmeansCentroidsDiv : 20
    - kmeansMaxDist : 29
    */

    int criterias[3];
    criterias[0] = 4 * warpSize;
    criterias[1] = regsPerBlock/32;
    criterias[2] = prop.maxThreadsPerMultiProcessor;

    int blockSize = min(min(criterias[0],criterias[1]), criterias[2]);

    blockSize = warpSize * ceil(blockSize/warpSize);

    return blockSize;
}